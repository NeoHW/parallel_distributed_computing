#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"
// #include <fstream>
// #include <chrono>


// Utility function to log timing results to a file
// void logTiming(const std::string& message, float time) {
//     std::ofstream logFile("timing_log.txt", std::ios_base::app);
//     logFile << message << ": " << time << " ms" << std::endl;
//     logFile.close();
// }

__global__ void matchKernel(
    const char* d_signature_seq,
    const int* d_signature_seq_starting_pos,
    const int* d_signature_seq_sizes,
    const char* d_sample_seq,
    const char* d_sample_qual,
    const int* d_sample_seq_starting_pos,
    const int* d_sample_seq_sizes,
    double* d_match_score,
    const int num_samples,
    const int num_signatures,
    const int total_pairs) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= total_pairs) return;

    // each thread takes one sample and one signature, scan whole sample for that signature
    int sample_idx = idx / num_signatures;
    int signature_idx = idx % num_signatures;

    int sample_start = d_sample_seq_starting_pos[sample_idx];
    int sample_length = d_sample_seq_sizes[sample_idx];
    const char* sample_seq = &d_sample_seq[sample_start];
    const char* sample_qual = &d_sample_qual[sample_start]; // same length for sample and qual string

    int signature_start = d_signature_seq_starting_pos[signature_idx];
    int signature_length = d_signature_seq_sizes[signature_idx];
    const char* signature_seq = &d_signature_seq[signature_start];

    double match_score = 0;
    int earliest_match = -1;

    // matching
    for (int i = 0; i <= sample_length - signature_length; ++i) {
        bool match = true;

        for (int j = 0; j < signature_length; ++j) {
            char sample_char = sample_seq[i + j];
            char signature_char = signature_seq[j];

            if (sample_char != 'N' && signature_char != 'N' && sample_char != signature_char) {
                match = false;
                break;
            }
        }
        
        if (match) {
            earliest_match = i;
            break;
        }
    }

    if (earliest_match != -1) {
        for (int i = 0; i < signature_length; ++i) {
            match_score += static_cast<int>(sample_qual[earliest_match + i]) - 33;
        }
        match_score = match_score / signature_length;
    }

    int output_idx = sample_idx * num_signatures + signature_idx;
    d_match_score[output_idx] = match_score;
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    // ### for profiling
    // hipEvent_t start, stop;
    // float elapsedTime;

    // Create CUDA events for timing
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    

    // hipError_t rc;

    //### Data Preparation
    int num_signatures = signatures.size();
    int num_samples = samples.size();

    // flatten signature sequence, have a vector containing start indexes and sizes for each string
    std::string flat_signature_seq;
    std::vector<int> signature_seq_starting_pos;
    std::vector<int> signature_seq_sizes;
    int sig_starting_pos = 0;

    int total_signature_size = 0;
    for (const auto& signature : signatures) {
        total_signature_size += signature.seq.size();
    }
    flat_signature_seq.reserve(total_signature_size);


    for (const auto& signature : signatures) {
        signature_seq_starting_pos.push_back(sig_starting_pos);
        flat_signature_seq += signature.seq;
        signature_seq_sizes.push_back(signature.seq.size());
        sig_starting_pos += signature.seq.size();
    }

    // flatten sample sequence, have vector containing start indexes and sizes for each string
    std::string flat_sample_seq;
    std::string flat_sample_qual;
    std::vector<int> sample_seq_starting_pos;
    std::vector<int> sample_seq_sizes;
    int sample_starting_pos = 0;

    int total_sample_size = 0;
    for (const auto& sample : samples) {
        total_sample_size += sample.seq.size();
    }
    flat_sample_seq.reserve(total_sample_size);
    flat_sample_qual.reserve(total_sample_size);

    for (const auto& sample : samples) {
        sample_seq_starting_pos.push_back(sample_starting_pos);
        flat_sample_seq += sample.seq;
        flat_sample_qual += sample.qual;
        sample_seq_sizes.push_back(sample.seq.size());
        sample_starting_pos += sample.seq.size();
    }

	int total_pairs = num_samples * num_signatures;
	int total_signature_seq_length = flat_signature_seq.size();
	int total_sample_seq_length = flat_sample_seq.size();

    // hipEventRecord(start); // profiling

	//### memory allocation and data transfer to device

	// for signature flattened sequence
	char* d_signature_seq;
	hipMalloc(&d_signature_seq, total_signature_seq_length * sizeof(char));
	hipMemcpy(d_signature_seq, flat_signature_seq.data(), total_signature_seq_length * sizeof(char), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy flat_signature_seq to device. Reason: %s\n", hipGetErrorString(rc));
    // }

	// for signature starting positions
	int* d_signature_seq_starting_pos;
	hipMalloc(&d_signature_seq_starting_pos, num_signatures * sizeof(int));
	hipMemcpy(d_signature_seq_starting_pos, signature_seq_starting_pos.data(), num_signatures * sizeof(int), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy signature_seq_starting_pos to device. Reason: %s\n", hipGetErrorString(rc));
    // }

	// for signature indivudal sizes
	int* d_signature_seq_sizes;
	hipMalloc(&d_signature_seq_sizes, num_signatures * sizeof(int));
	hipMemcpy(d_signature_seq_sizes, signature_seq_sizes.data(), num_signatures * sizeof(int), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy signature_seq_sizes to device. Reason: %s\n", hipGetErrorString(rc));
    // }

	// for sample flattened sequence
	char* d_sample_seq;
    hipMalloc(&d_sample_seq, total_sample_seq_length * sizeof(char));
    hipMemcpy(d_sample_seq, flat_sample_seq.data(), total_sample_seq_length * sizeof(char), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy sample_seq to device. Reason: %s\n", hipGetErrorString(rc));
    // }

    // for sample flattened quality string
	char* d_sample_qual;
    hipMalloc(&d_sample_qual, total_sample_seq_length * sizeof(char));
    hipMemcpy(d_sample_qual, flat_sample_qual.data(), total_sample_seq_length * sizeof(char), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy sample_qual to device. Reason: %s\n", hipGetErrorString(rc));
    // }

	// for sample starting positions
	int* d_sample_seq_starting_pos;
	hipMalloc(&d_sample_seq_starting_pos, num_samples * sizeof(int));
	hipMemcpy(d_sample_seq_starting_pos, sample_seq_starting_pos.data(), num_samples * sizeof(int), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy sample_seq_starting_pos to device. Reason: %s\n", hipGetErrorString(rc));
    // }

    // for sample indivudal sizes
	int* d_sample_seq_sizes;
	hipMalloc(&d_sample_seq_sizes, num_samples * sizeof(int));
	hipMemcpy(d_sample_seq_sizes, sample_seq_sizes.data(), num_samples * sizeof(int), hipMemcpyHostToDevice);
	// if (rc != hipSuccess) {
    //     printf("Could not copy sample_seq_sizes to device. Reason: %s\n", hipGetErrorString(rc));
    // }

    // for storing match score on device
    double* d_match_score;
    hipMalloc(&d_match_score, total_pairs * sizeof(double));


    // for profiling
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // logTiming("Data Transfer to Device", elapsedTime);

    // hipEventRecord(start);

	//### Kernel Execution
    int THREADS_PER_BLOCK = 256; // try 256,512,1024
    // const dim3 threadsPerBlock = { THREADS_PER_BLOCK, 1, 1 }; 
    const int blocks_needed = (total_pairs + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // const dim3 numBlocks = { blocks_needed, 1, 1 };

    matchKernel<<<blocks_needed, THREADS_PER_BLOCK>>>(
        d_signature_seq,
        d_signature_seq_starting_pos,
        d_signature_seq_sizes,
        d_sample_seq,
        d_sample_qual,
        d_sample_seq_starting_pos,
        d_sample_seq_sizes,
        d_match_score,
        num_samples,
        num_signatures,
        total_pairs);

    hipDeviceSynchronize();

    // for profiling
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // logTiming("Kernel Execution", elapsedTime);

    // hipEventRecord(start);

    // ### Result Collection

    std::vector<double> h_match_score(total_pairs);
    hipMemcpy(h_match_score.data(), d_match_score, total_pairs * sizeof(double), hipMemcpyDeviceToHost);

    for (int sample_idx = 0; sample_idx < num_samples; ++sample_idx) {
        for (int signature_idx = 0; signature_idx < num_signatures; ++signature_idx) {
            int idx = sample_idx * num_signatures + signature_idx;
            if (h_match_score[idx] != 0) {
                MatchResult result = {
                    samples[sample_idx].name,
                    signatures[signature_idx].name,
                    h_match_score[idx]
                };
                matches.push_back(result);
            }
        }
    }

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // logTiming("Result Collection", elapsedTime);

    // ### Cleanup
    hipFree(d_signature_seq);
    hipFree(d_signature_seq_starting_pos);
    hipFree(d_signature_seq_sizes);
    hipFree(d_sample_seq);
    hipFree(d_sample_qual);
    hipFree(d_sample_seq_starting_pos);
    hipFree(d_sample_seq_sizes);
    hipFree(d_match_score);

    // hipEventDestroy(start);
    // hipEventDestroy(stop);
}
